#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <custatevec.h>

int main() {
    // Initialize CUDA and cuStateVec
    hipSetDevice(0);
    custatevecHandle_t handle;
    custatevecCreate(&handle);

    std::cout << "Simple cuQuantum Example: Creating a Bell State" << std::endl;

    // Create a 2-qubit system
    const int numQubits = 2;
    const size_t dim = 1ULL << numQubits;  // 2^numQubits = 4

    // Allocate device memory for state vector
    hipDoubleComplex* d_sv;
    hipMalloc((void**)&d_sv, dim * sizeof(hipDoubleComplex));

    // Initialize to |00⟩ state
    hipDoubleComplex one = {1.0, 0.0};
    hipDoubleComplex zero = {0.0, 0.0};

    hipMemcpy(&d_sv[0], &one, sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    for (int i = 1; i < dim; i++) {
        hipMemcpy(&d_sv[i], &zero, sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    }

    // Apply Hadamard to first qubit
    const double factor = 1.0 / std::sqrt(2.0);
    hipDoubleComplex hadamard[4] = {
        {factor, 0.0}, {factor, 0.0},
        {factor, 0.0}, {-factor, 0.0}
    };

    // Target qubit array - first qubit (index 0)
    const int32_t targets[1] = {0};
    const int32_t adjoint = 0;  // Not using adjoint (conjugate transpose)

    // CORRECTED: Apply Hadamard with proper parameters
    custatevecApplyMatrix(
        handle,
        d_sv,
        HIP_C_64F,
        numQubits,
        hadamard,
        HIP_C_64F,              // Matrix data type
        CUSTATEVEC_MATRIX_LAYOUT_ROW,
        adjoint,                 // Not using adjoint
        targets,                 // Array of target qubits
        1,                       // Number of target qubits
        nullptr,                 // No control qubits
        nullptr,                 // No control bit values
        0,                       // Number of control qubits
        CUSTATEVEC_COMPUTE_64F,
        nullptr,
        0
    );

    // Apply CNOT using a matrix-based approach instead of custatevecApplyCNOT
    // CNOT matrix: [[1,0,0,0], [0,1,0,0], [0,0,0,1], [0,0,1,0]]
    hipDoubleComplex cnotMatrix[16] = {
        {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},  // First row
        {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},  // Second row
        {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0},  // Third row
        {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}   // Fourth row
    };

    // Apply to both qubits
    const int32_t twoQubits[2] = {0, 1};  // Control qubit 0, target qubit 1

    custatevecApplyMatrix(
        handle,
        d_sv,
        HIP_C_64F,
        numQubits,
        cnotMatrix,
        HIP_C_64F,                 // Matrix data type
        CUSTATEVEC_MATRIX_LAYOUT_ROW,
        adjoint,                    // Not using adjoint
        twoQubits,                  // Array of target qubits (both qubits)
        2,                          // Number of target qubits (2 qubits)
        nullptr,                    // No additional control qubits
        nullptr,                    // No control bit values
        0,                          // Number of additional control qubits
        CUSTATEVEC_COMPUTE_64F,
        nullptr,
        0
    );

    // Copy the result back to examine
    std::vector<hipDoubleComplex> h_result(dim);
    hipMemcpy(h_result.data(), d_sv, dim * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // Print the state vector
    std::cout << "Final quantum state (Bell state):" << std::endl;
    std::cout << "|00⟩: " << h_result[0].x << " + " << h_result[0].y << "i" << std::endl;
    std::cout << "|01⟩: " << h_result[1].x << " + " << h_result[1].y << "i" << std::endl;
    std::cout << "|10⟩: " << h_result[2].x << " + " << h_result[2].y << "i" << std::endl;
    std::cout << "|11⟩: " << h_result[3].x << " + " << h_result[3].y << "i" << std::endl;

    // Expected result: 1/sqrt(2) * (|00⟩ + |11⟩)

    // Clean up
    hipFree(d_sv);
    custatevecDestroy(handle);

    return 0;
}